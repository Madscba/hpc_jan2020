extern "C" {

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#define BLOCK_SIZE 16
__global__ void kernel_gpu5(int m, int n, int k, double *A, double *B, double *C){
		//Block
		int y = blockIdx.y; //Row
      int x = blockIdx.x; //Col
      //Thread
      int i = threadIdx.y; //Row
    	int j = threadIdx.x; //Col

      double tmp = 0;
      double *C_sub = &C[y*n*BLOCK_SIZE + x*BLOCK_SIZE];


      for (int l=0; l< (k/BLOCK_SIZE); l++){

         double *A_sub = &A[l*BLOCK_SIZE + k*BLOCK_SIZE * y];
         double *B_sub = &B[l*BLOCK_SIZE*n + BLOCK_SIZE * x];
         __shared__ double tmp_A[BLOCK_SIZE][BLOCK_SIZE];
         __shared__ double tmp_B[BLOCK_SIZE][BLOCK_SIZE];
         tmp_A[i][j] = A_sub[i*k + j ];
         tmp_B[i][j] = B_sub[i*n + j];

         // Synchronize to make sure the sub-matrices are loaded
         // before starting the computation
         __syncthreads();

         for (int subM = 0; subM<BLOCK_SIZE; subM++) {
			   tmp += tmp_A[i][subM]*tmp_B[subM][j];
		   }
		   // Sync before moving on to new submatrices
        	__syncthreads();
      }
      C_sub[i*n + j] = tmp;
}
void matmult_gpu5(int m, int n, int k, double *A_h, double *B_h, double *C_h) 
   { 
      // Allocate A_d, B_d, C_d and transfer data 
      //Initialize variables
      double *A_d, *B_d, *C_d;
      int A_size = m*k*sizeof( double );
      int B_size = k*n*sizeof( double );
      int C_size = m*n*sizeof( double );
      // Allocate on device
      hipMalloc( (void**)&A_d, A_size );
      hipMalloc( (void**)&B_d, B_size );
      hipMalloc( (void**)&C_d, C_size );
      //Copy to device from host
      hipMemcpy(A_d, A_h, A_size, hipMemcpyHostToDevice);
      hipMemcpy(B_d, B_h, B_size, hipMemcpyHostToDevice);

      //Define block grid
      dim3 block(BLOCK_SIZE,BLOCK_SIZE);
      dim3 grid(ceil((double) n/BLOCK_SIZE),  ceil((double) m/BLOCK_SIZE));
      // Launch kernel and synchronize
      kernel_gpu5<<<grid,block>>>(m,n,k,A_d,B_d,C_d);
      hipDeviceSynchronize();

      hipMemcpy(C_h, C_d, C_size, hipMemcpyDeviceToHost);
      
      // Free A_d, B_d, C_d
      hipFree(A_d);
      hipFree(B_d);
      hipFree(C_d);
   } 
}