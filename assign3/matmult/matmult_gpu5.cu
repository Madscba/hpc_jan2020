extern "C" {

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
__global__ void kernel_gpu5(int m, int n, int k, double *A, double *B, double *C){
		int x = blockIdx.x;
		int y = blockIdx.y;
      double tmp = 0;
}
void matmult_gpu5(int m, int n, int k, double *A_h, double *B_h, double *C_h) 
   { 
      // Allocate A_d, B_d, C_d and transfer data 
      //Initialize variables
      double *A_d, *B_d, *C_d;
      int A_size = m*k*sizeof( double );
      int B_size = k*n*sizeof( double );
      int C_size = m*n*sizeof( double );
      // Allocate on device
      hipMalloc( (void**)&A_d, A_size );
      hipMalloc( (void**)&B_d, B_size );
      hipMalloc( (void**)&C_d, C_size );
      hipMemset(C_d, 0, C_size);
      //Copy to device from host
      hipMemcpy(A_d, A_h, A_size, hipMemcpyHostToDevice);
      hipMemcpy(B_d, B_h, B_size, hipMemcpyHostToDevice);

      const int BLOCK_SIZE = 16;
      //Define block grid
      dim3 block(BLOCK_SIZE,BLOCK_SIZE);
      dim3 grid(ceil((double) n/BLOCK_SIZE),  ceil((double) m/BLOCK_SIZE));
      // Launch kernel and synchronize
      kernel_gpu5<<<grid,block>>>(m,n,k,A_d,B_d,C_d);
      hipDeviceSynchronize();

      hipMemcpy(C_h, C_d, C_size, hipMemcpyDeviceToHost);
      
      // Free A_d, B_d, C_d
      hipFree(A_d);
      hipFree(B_d);
      hipFree(C_d);
   } 
}