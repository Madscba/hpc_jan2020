extern "C" {

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#define BLOCK_SIZE 16
__global__ void kernel_gpu5(int m, int n, int k, double *A, double *B, double *C){
		//Block
		int y = blockIdx.y; //Row
      int x = blockIdx.x; //Col
      //Thread
      int i = threadIdx.y; //Row
    	int j = threadIdx.x; //Col

      double tmp = 0;

      __shared__ double tmp_A[BLOCK_SIZE][BLOCK_SIZE];
      __shared__ double tmp_B[BLOCK_SIZE][BLOCK_SIZE];

      for (int l=0; l<k/BLOCK_SIZE; l++){
         tmp_A[i][j] = A[i*k + j + l*BLOCK_SIZE + k*BLOCK_SIZE * y];
         tmp_B[i][j] = B[i*n + j + l*BLOCK_SIZE*n + BLOCK_SIZE * x];

         // Synchronize to make sure the sub-matrices are loaded
         // before starting the computation
         __syncthreads();

         for (int subM = 0; subM<BLOCK_SIZE; subM++) {
			   tmp += tmp_A[i][subM]*tmp_B[subM][j];
		   }
		   // Sync before moving on to new submatrices
        	__syncthreads();
      }
      C[i*n + j + y*n*BLOCK_SIZE + x*BLOCK_SIZE] = tmp;
}
void matmult_gpu5(int m, int n, int k, double *A_h, double *B_h, double *C_h) 
   { 
      // Allocate A_d, B_d, C_d and transfer data 
      //Initialize variables
      double *A_d, *B_d, *C_d;
      int A_size = m*k*sizeof( double );
      int B_size = k*n*sizeof( double );
      int C_size = m*n*sizeof( double );
      // Allocate on device
      hipMalloc( (void**)&A_d, A_size );
      hipMalloc( (void**)&B_d, B_size );
      hipMalloc( (void**)&C_d, C_size );
      hipMemset(C_d, 0, C_size);
      //Copy to device from host
      hipMemcpy(A_d, A_h, A_size, hipMemcpyHostToDevice);
      hipMemcpy(B_d, B_h, B_size, hipMemcpyHostToDevice);

      //Define block grid
      dim3 block(BLOCK_SIZE,BLOCK_SIZE);
      dim3 grid(ceil((double) n/BLOCK_SIZE),  ceil((double) m/BLOCK_SIZE));
      // Launch kernel and synchronize
      kernel_gpu5<<<grid,block>>>(m,n,k,A_d,B_d,C_d);
      hipDeviceSynchronize();

      hipMemcpy(C_h, C_d, C_size, hipMemcpyDeviceToHost);
      
      // Free A_d, B_d, C_d
      hipFree(A_d);
      hipFree(B_d);
      hipFree(C_d);
   } 
}