extern "C" {

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
__global__ void kernel_gpu4(int m, int n, int k, double *A, double *B, double *C){
		int ti = blockIdx.y * blockDim.y + threadIdx.y;
		int tj = blockIdx.x * blockDim.x + threadIdx.x;
      double tmp11 = 0;
      double tmp12 = 0;
      double tmp21 = 0;
      double tmp22 = 0;
      double tmp_a1, tmp_a2, tmp_b1, tmp_b2;
      //printf("i = %d, j = %d \n",ti,tj);
      if (2*ti < m && 2*tj < n){
         for (int l = 0; l < k; l++)
         {
            tmp_a1 = A[2*ti*k + l];
            tmp_a2 = A[(2*ti+1)*k + l];
            tmp_b1 = B[l*n + 2*tj];
            tmp_b2 = B[l*n + 2*tj+1];
            tmp11 += tmp_a1 * tmp_b1;
            tmp12 += tmp_a1 * tmp_b2;
            tmp21 += tmp_a2 * tmp_b1;
            tmp22 += tmp_a2 * tmp_b2;
         }
         C[2*ti*n+tj*2] = tmp11;
         C[2*ti*n+tj*2+1] = tmp12;
         C[(2*ti+1)*n+tj*2] = tmp21;
         C[(2*ti+1)*n+tj*2+1] = tmp22;
         }
			
}
void matmult_gpu4(int m, int n, int k, double *A_h, double *B_h, double *C_h) 
   { 
      // Allocate A_d, B_d, C_d and transfer data 
      //Initialize variables
      double *A_d, *B_d, *C_d;
      int A_size = m*k*sizeof( double );
      int B_size = k*n*sizeof( double );
      int C_size = m*n*sizeof( double );
      // Allocate on device
      hipMalloc( (void**)&A_d, A_size );
      hipMalloc( (void**)&B_d, B_size );
      hipMalloc( (void**)&C_d, C_size );
      hipMemset(C_d, 0, C_size);
      //Copy to device from host
      hipMemcpy(A_d, A_h, A_size, hipMemcpyHostToDevice);
      hipMemcpy(B_d, B_h, B_size, hipMemcpyHostToDevice);

      const int BLOCK_SIZE = 16;
      //Define block grid
      dim3 block(BLOCK_SIZE/2,BLOCK_SIZE/2);
      dim3 grid(ceil((double) n/BLOCK_SIZE),  ceil((double) m/BLOCK_SIZE));
      // Launch kernel and synchronize
      kernel_gpu4<<<grid,block>>>(m,n,k,A_d,B_d,C_d);
      hipDeviceSynchronize();

      hipMemcpy(C_h, C_d, C_size, hipMemcpyDeviceToHost);
      
      // Free A_d, B_d, C_d
      hipFree(A_d);
      hipFree(B_d);
      hipFree(C_d);
   } 
}