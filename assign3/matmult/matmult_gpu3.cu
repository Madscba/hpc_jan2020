extern "C" {

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
__global__ void kernel_gpu3(int m, int n, int k, double *A, double *B, double *C){
		int ti = blockIdx.y * blockDim.y + threadIdx.y;
		int tj = blockIdx.x * blockDim.x + threadIdx.x;
      double tmp1 = 0;
      double tmp2 = 0;
      double tmp_b;
      //printf("i = %d, j = %d \n",ti,tj);
      if (2*ti < m && tj < n){
         for (int l = 0; l < k; l++)
         {
            tmp_b = B[l*n + tj];
            tmp1 += A[2*ti*k + l] * tmp_b;
            tmp2 += A[(2*ti+1)*k + l] * tmp_b;
         }
         C[2*ti*n+tj] = tmp1;
         C[(2*ti+1)*n+tj] = tmp2;
         }
			
}
void matmult_gpu3(int m, int n, int k, double *A_h, double *B_h, double *C_h) 
   { 
      // Allocate A_d, B_d, C_d and transfer data 
      //Initialize variables
      double *A_d, *B_d, *C_d;
      int A_size = m*k*sizeof( double );
      int B_size = k*n*sizeof( double );
      int C_size = m*n*sizeof( double );
      // Allocate on device
      hipMalloc( (void**)&A_d, A_size );
      hipMalloc( (void**)&B_d, B_size );
      hipMalloc( (void**)&C_d, C_size );
      hipMemset(C_d, 0, C_size);
      //Copy to device from host
      hipMemcpy(A_d, A_h, A_size, hipMemcpyHostToDevice);
      hipMemcpy(B_d, B_h, B_size, hipMemcpyHostToDevice);

      const int BLOCK_SIZE = 16;
      //Define block grid
      dim3 block(BLOCK_SIZE,BLOCK_SIZE/2);
      dim3 grid(ceil((double) n/BLOCK_SIZE),  ceil((double) m/BLOCK_SIZE));
      // Launch kernel and synchronize
      kernel_gpu3<<<grid,block>>>(m,n,k,A_d,B_d,C_d);
      hipDeviceSynchronize();

      hipMemcpy(C_h, C_d, C_size, hipMemcpyDeviceToHost);
      
      // Free A_d, B_d, C_d
      hipFree(A_d);
      hipFree(B_d);
      hipFree(C_d);
   } 
}