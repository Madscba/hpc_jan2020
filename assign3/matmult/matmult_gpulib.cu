
#include <hip/hip_runtime.h>
#include <hipblas.h>
extern "C" {


void matmult_gpulib(int m,int n,int k,double *A_h,double *B_h,double *C_h){
	hipblasStatus_t stat; 
	hipblasHandle_t handle; 
	stat = hipblasCreate(&handle);


	// Allocate A_d, B_d, C_d and transfer data 
      //Initialize variables
      double *A_d, *B_d, *C_d;
      int A_size = m*k*sizeof( double );
      int B_size = k*n*sizeof( double );
      int C_size = m*n*sizeof( double );
      // Allocate on device
      hipMalloc( (void**)&A_d, A_size );
      hipMalloc( (void**)&B_d, B_size );
      hipMalloc( (void**)&C_d, C_size );

      //Copy to device from host
      hipMemcpy(A_d, A_h, A_size, hipMemcpyHostToDevice);
      hipMemcpy(B_d, B_h, B_size, hipMemcpyHostToDevice);
      hipMemcpy(C_d, C_h, C_size, hipMemcpyHostToDevice);
	  double alf = 1;
	  double bet = 0;
	  const double *alpha = &alf;
	  const double *beta = &bet;

	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, alpha, B_d, n, A_d, k, beta, C_d,n);

	hipMemcpy(C_h, C_d, C_size, hipMemcpyDeviceToHost);
      
	// Free A_d, B_d, C_d
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
	
}
}
