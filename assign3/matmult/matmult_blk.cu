
#include <hip/hip_runtime.h>
extern "C" {
#define MIN(x, y) (((x) < (y)) ? (x) : (y))
void
matmult_blk(int m, int n, int k, double *A, double *B, double *C, int bs) {
  int i,j, i1, i2, j1,j2, l1, l2, imin, lmin,jmin;

  /* initializing C to be zero */
  for ( i = 0; i < m; i++ ) {
      for ( j = 0; j < n; j++ ) {
        C[i*n+j] = 0;
      }
    }


  /* matrix multiplication */
  for ( i1 = 0; i1 < m; i1+=bs ) {
    imin = MIN(m-i1, bs);
    for ( l1 = 0; l1 < k; l1+=bs) {
      lmin = MIN(k-l1, bs);
      for ( j1 = 0; j1 < n; j1+=bs ) {
        jmin = MIN(n-j1,bs);
        for ( i2 = 0; i2 < imin; i2++){
          for ( l2=0; l2 < lmin; l2++){
            for( j2 = 0; j2 < jmin; j2++){
              C[(i1+i2)*n+j1+j2] += A[(i1+i2)*k+l1+l2]*B[(l1+l2)*n+j1+j2];
              }
            }
          }
        }
      }
    }
  }
}