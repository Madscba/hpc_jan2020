
#include <hip/hip_runtime.h>

extern "C" {
__global__
void kernel_gpu1(int m, int n, int k, double *A, double *B, double *C){

	for(int i = 0; i< m; i++)
	{	
		for(int l = 0; l < k; l++)
		{
			for(int j = 0; j < n; j++)
			{
				C[i*n+j] += A[i*k+l] * B[l*n+j];
			}
			
		}
	}
}
void matmult_gpu1(int m, int n, int k, double *A_h, double *B_h, double *C_h) 
   { 
      // Allocate A_d, B_d, C_d and transfer data 
      //Initialize variables
      double *A_d, *B_d, *C_d;
      int A_size = m*k*sizeof( double );
      int B_size = k*n*sizeof( double );
      int C_size = m*n*sizeof( double );
        for(int i = 0; i< m; i++)
        {
          for(int j = 0; j < n; j++)
          {
            C_h[i*n+j] = 0;
          }
        }

      // Allocate on device
      hipMalloc( (void**)&A_d, A_size );
      hipMalloc( (void**)&B_d, B_size );
      hipMalloc( (void**)&C_d, C_size );

      //Copy to device from host
      hipMemcpy(A_d, A_h, A_size, hipMemcpyHostToDevice);
      hipMemcpy(B_d, B_h, B_size, hipMemcpyHostToDevice);
      hipMemcpy(C_d, C_h, C_size, hipMemcpyHostToDevice);
      // Launch kernel and synchronize 
      kernel_gpu1<<<1,1>>>(m,n,k,A_d,B_d,C_d);
      hipDeviceSynchronize();

      hipMemcpy(C_h, C_d, C_size, hipMemcpyDeviceToHost);
      
      // Free A_d, B_d, C_d
      hipFree(A_d);
      hipFree(B_d);
      hipFree(C_d);
   } 
}