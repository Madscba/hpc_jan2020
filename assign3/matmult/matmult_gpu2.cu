
#include <hip/hip_runtime.h>
#include <math.h>
#define BLOCK_SIZE 256
extern "C" {
__global__ void kernel_gpu2(int m, int n, int k, double *A, double *B, double *C){
		int ti = blockIdx.y * blockDim.y + threadIdx.y;
		int tj = blockIdx.x * blockDim.x + threadIdx.x;
      double tmp = 0;

      if (ti < m && tj < n){
         for (int l = 0; l < k; l++)
         {
            tmp += A[ti*k + l] * B[l*n + tj];
         }
         C[ti*n+tj] = tmp;
         }
			
}
void matmult_gpu2(int m, int n, int k, double *A_h, double *B_h, double *C_h) 
   { 
      // Allocate A_d, B_d, C_d and transfer data 
      //Initialize variables
      double *A_d, *B_d, *C_d;
      int A_size = m*k*sizeof( double );
      int B_size = k*n*sizeof( double );
      int C_size = m*n*sizeof( double );
      // Allocate on device
      hipMalloc( (void**)&A_d, A_size );
      hipMalloc( (void**)&B_d, B_size );
      hipMalloc( (void**)&C_d, C_size );
      hipMemset(C_d, 0, C_size);
      //Copy to device from host
      hipMemcpy(A_d, A_h, A_size, hipMemcpyHostToDevice);
      hipMemcpy(B_d, B_h, B_size, hipMemcpyHostToDevice);

      
      //Define block grid
      dim3 block(BLOCK_SIZE,BLOCK_SIZE);
      dim3 grid(n/BLOCK_SIZE, m/BLOCK_SIZE);
      // Launch kernel and synchronize
      kernel_gpu2<<<grid,block>>>(m,n,k,A_d,B_d,C_d);
      hipDeviceSynchronize();

      hipMemcpy(C_h, C_d, C_size, hipMemcpyDeviceToHost);
      
      // Free A_d, B_d, C_d
      hipFree(A_d);
      hipFree(B_d);
      hipFree(C_d);
   } 
}