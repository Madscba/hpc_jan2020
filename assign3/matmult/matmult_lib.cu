extern "C" {

#include <hip/hip_runtime.h>
#include <cblas.h>


void matmult_lib(int m,int n,int k,double *A,double *B,double *C){
	/* Level 3 BLAS: cblas_interface.pdf, page 191 line 5
	 * performed operation: C <- alpha * AB + beta*C */
	int alpha = 1;
	int beta = 0;
	//int CblasNoTrans = 111;
	int lda = k;
	int ldb = n;
	int ldc = n;
	//DGEMM(TRANSA, TRANSB, m, n, k, ALPHA, A, LDA, B, LDB, BETA, C, LDC);
	cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m,n,k, alpha, A, lda, B, ldb, beta, C, ldc);
	
}
}
