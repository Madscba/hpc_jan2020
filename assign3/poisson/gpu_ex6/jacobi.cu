/* jacobi.c - Poisson problem in 3d
 * 
 */

#include <hip/hip_runtime.h>
#include <math.h>
#include <float.h>

__global__
void
jacobi(double ***u, double ***u_old, double ***f, int N, double delta) {
    int i, j, k;
	double tmpi, tmpj, tmpk;
	i = blockIdx.x * blockDim.x * gridDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y * gridDim.y  + threadIdx.y;
	k = blockIdx.z * blockDim.z * gridDim.z  + threadIdx.z;
	tmpi = (u_old[i-1][j][k] + u_old[i+1][j][k]);
	tmpj = (u_old[i][j-1][k] + u_old[i][j+1][k]);
	tmpk = (u_old[i][j][k-1] + u_old[i][j][k+1]);
	u[i][j][k] = (tmpi + tmpj + tmpk + delta*f[i][j][k]) / 6.0;

}
