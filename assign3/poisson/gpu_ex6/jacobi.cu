#include "hip/hip_runtime.h"
/* jacobi.c - Poisson problem in 3d
 * 
 */
#include <math.h>
#include <float.h>
#include <hip/hip_runtime_api.h>
#include "transfer3d_gpu.h"
#include "frobenius.h"

__global__
void
jacobi_kernel(double ***u, double ***u_old, double ***f, int N, double delta) {
    int i, j, k;
	double tmpi, tmpj, tmpk;
	i = blockIdx.z * blockDim.z + threadIdx.z+1;
	j = blockIdx.y * blockDim.y + threadIdx.y+1;
	k = blockIdx.x * blockDim.x + threadIdx.x+1;
	if (i < N+1 && j < N+1 && k < N+1){
		tmpi = (u_old[i-1][j][k] + u_old[i+1][j][k]);
		tmpj = (u_old[i][j-1][k] + u_old[i][j+1][k]);
		tmpk = (u_old[i][j][k-1] + u_old[i][j][k+1]);
		u[i][j][k] = (tmpi + tmpj + tmpk + delta*f[i][j][k]) / 6.0;
	}
}

int
jacobi(double ***u_d, double ***u_old_d, double ***f_d, double ***u_h, double ***u_old_h, double ***f_h, int N, double delta, int iter_max, int NUM_BLOCKS, int THREADS_PER_BLOCK) {
	double*** temp;
	int k = 0;
    double d = 0.0;
    dim3 dimBlock(NUM_BLOCKS,THREADS_PER_BLOCK,1);
    dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y,1); 
	while(k<iter_max)
    {
        // Execute kernel function
		jacobi_kernel<<<dimGrid,dimBlock>>>(u_d,u_old_d,f_d,N,delta);        
		checkCudaErrors(hipDeviceSynchronize());
		//  #Comment out when benchmarking!!#
        if ((k % 100) == 0)
		{   
			transfer_3d(u_h,u_d,N+2,N+2,N+2,hipMemcpyDeviceToHost);
			transfer_3d(u_old_h,u_old_d,N+2,N+2,N+2,hipMemcpyDeviceToHost);
            d = frobenius(u_h,u_old_h,N);
			printf("%i  %.5f\n", k, d);
        }
        //  #Comment out when benchmarking!!#
        temp = u_old_d;
        u_old_d = u_d;
        u_d  = temp;
        k+=1;
    }
	return k;
}
