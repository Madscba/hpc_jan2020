
#include <hip/hip_runtime.h>
#include <math.h>
__global__
void frobenius_kernel(double ***u, double ***u_old, int N,double sum){
    int i, j, k;
    double dist;
    for (i = 1; i < N+1; i++) 
	{
		for (j = 1; j < N+1; j++)   
		{
			for (k = 1; k < N+1; k++) 
			{   
                dist = u[i][j][k] - u_old[i][j][k];
                sum += dist*dist;
            }
        }
    }
}

double frobenius(double ***u, double ***u_old, int N){
    int i, j, k;
    double dist;
    double sum = 0.0;
    for (i = 1; i < N+1; i++) 
	{
		for (j = 1; j < N+1; j++)   
		{
			for (k = 1; k < N+1; k++) 
			{   
                dist = u[i][j][k] - u_old[i][j][k];
                sum += dist*dist;
            }
        }
    }
    return sqrt(sum);
}