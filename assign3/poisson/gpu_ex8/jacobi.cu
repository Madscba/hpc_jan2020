#include "hip/hip_runtime.h"
/* jacobi.c - Poisson problem in 3d
 * 
 */
#include <math.h>
#include <float.h>
#include <hip/hip_runtime_api.h>
#include "transfer3d_gpu.h"
#include "frobenius.h"


__global__ 
void
jacobi_reduction_baseline(double ***u, double ***u_old, double ***f, int N, double delta, double *d) {
	double a;
	int i = blockIdx.z * blockDim.z + threadIdx.z+1;
	int j = blockIdx.y * blockDim.y + threadIdx.y+1;
	int k = blockIdx.x * blockDim.x + threadIdx.x+1; 
    if (i < N+1 && j < N+1 && k < N+1) {
		double tmpi = (u_old[i-1][j][k] + u_old[i+1][j][k]);
		double tmpj = (u_old[i][j-1][k] + u_old[i][j+1][k]);
		double tmpk = (u_old[i][j][k-1] + u_old[i][j][k+1]);
		u[i][j][k] = (tmpi + tmpj + tmpk + delta*f[i][j][k]) / 6.0;
		a = sqrt((u[i][j][k]-u_old[i][j][k])*(u[i][j][k]-u_old[i][j][k]));
        atomicAdd(d,a);
    }
}

__inline__ __device__ 
double warpReduceSum(double value) { 
    for (int i = 16; i > 0; i /= 2) 
        value += __shfl_down_sync(-1, value, i);  
    return value; 
}

__global__ 
void jacobi_reduction_warp(double ***u, double ***u_old, double ***f, int N, double delta, double *d) 
{ 
	double value;
    int i = blockIdx.z * blockDim.z + threadIdx.z+1;
	int j = blockIdx.y * blockDim.y + threadIdx.y+1;
	int k = blockIdx.x * blockDim.x + threadIdx.x+1;
	if (i < N+1 && j < N+1 && k < N+1) {
		double tmpi = (u_old[i-1][j][k] + u_old[i+1][j][k]);
		double tmpj = (u_old[i][j-1][k] + u_old[i][j+1][k]);
		double tmpk = (u_old[i][j][k-1] + u_old[i][j][k+1]);
		u[i][j][k] = (tmpi + tmpj + tmpk + delta*f[i][j][k]) / 6.0;
		value = sqrt((u[i][j][k]-u_old[i][j][k])*(u[i][j][k]-u_old[i][j][k]));
	} else {
		value = 0.0;
	}
    value = warpReduceSum(value); 
    if (threadIdx.x % warpSize == 0 && threadIdx.y % warpSize == 0 && threadIdx.z % warpSize == 0){
		 atomicAdd(d, value);
	} 
}

int
jacobi(double ***u_d, double ***u_old_d, double ***f_d, double ***u_h, double ***u_old_h, double ***f_h, int N, double delta, int iter_max, int NUM_BLOCKS, int THREADS_PER_BLOCK) {
	double*** temp;
	int k = 0;
	double frob;
    double *d;
	double *d_h0;
	double *d_h1;
	hipHostMalloc((void**)&d_h0, sizeof(double)*1);
	hipHostMalloc((void**)&d_h1, sizeof(double)*1);
	d_h1[0] = 1000000.0;	
	hipMalloc((void**)&d, sizeof(double)*1);
	double tolerance = 1.0; 
	dim3 dimBlock(8,8,8); //Threads per block
    dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y,(N+dimBlock.z-1)/dimBlock.z); // Blocks in grid
	while(d_h1[0]>tolerance && k<iter_max)
    {
		d_h0[0] = 0.0;
		hipMemcpy(d,d_h0, sizeof(double), hipMemcpyHostToDevice);
        // Execute kernel function
		jacobi_reduction_warp<<<dimGrid,dimBlock>>>(u_d,u_old_d,f_d,N,delta,d);        
		checkCudaErrors(hipDeviceSynchronize());
		hipMemcpy(d_h0,d, sizeof(double)*1, hipMemcpyDeviceToHost);
		d_h1[0] = d_h0[0];
        temp = u_old_d;
        u_old_d = u_d;
        u_d  = temp;
        k+=1;
    }
	return k;
}