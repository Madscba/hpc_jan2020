#include "hip/hip_runtime.h"
/* jacobi.c - Poisson problem in 3d
 * 
 */
#include <math.h>
#
#include <float.h>
#include <hip/hip_runtime_api.h>
#include "transfer3d_gpu.h"
#include "frobenius.h"


__global__ 
void
jacobi_reduction_baseline(double ***u, double ***u_old, double ***f, int N, double delta, double *d) {
    double *a;
	i = blockIdx.z * blockDim.z + threadIdx.z+1;
	j = blockIdx.y * blockDim.y + threadIdx.y+1;
	k = blockIdx.x * blockDim.x + threadIdx.x+1; 
    if (i < N+1 && j < N+1 && k < N+1) {
		tmpi = (u_old[i-1][j][k] + u_old[i+1][j][k]);
		tmpj = (u_old[i][j-1][k] + u_old[i][j+1][k]);
		tmpk = (u_old[i][j][k-1] + u_old[i][j][k+1]);
		u[i][j][k] = (tmpi + tmpj + tmpk + delta*f[i][j][k]) / 6.0;
		a = sqrt((u[i][j][k]-u_old[i][j][k])*(u[i][j][k]-u_old[i][j][k]));
        atomicAdd(d,a);
    }
}


__inline__ __device__ 
double blockReduceSum(double*** value) { 
    __shared__ double smem[32]; // Max 32 warp sums 
 
    if (threadIdx.x < warpSize) 
        smem[threadIdx.x] = 0; 
    __syncthreads(); 
 
    value = warpReduceSum(value); 
 
    if (threadIdx.x % warpSize == 0) 
        smem[threadIdx.x / warpSize] = value; 
    __syncthreads(); 
 
    if (threadIdx.x < warpSize) 
        value = smem[threadIdx.x]; 
    return warpReduceSum(value); 
}

__inline__ __device__ 
double warpReduceSum(double value) { 
    for (int i = 16; i > 0; i /= 2) 
        value += __shfl_down_sync(-1, value, i);  
    return value; 
}

__global__ 
void jacobi_reduction_warp(double ***u, double ***u_old, double ***f, int N, double delta, double *d) 
{ 
	double value;
    i = blockIdx.z * blockDim.z + threadIdx.z+1;
	j = blockIdx.y * blockDim.y + threadIdx.y+1;
	k = blockIdx.x * blockDim.x + threadIdx.x+1;
	if (i < N+1 && j < N+1 && k < N+1) {
		tmpi = (u_old[i-1][j][k] + u_old[i+1][j][k]);
		tmpj = (u_old[i][j-1][k] + u_old[i][j+1][k]);
		tmpk = (u_old[i][j][k-1] + u_old[i][j][k+1]);
		u[i][j][k] = (tmpi + tmpj + tmpk + delta*f[i][j][k]) / 6.0;
		value = sqrt((u[i][j][k]-u_old[i][j][k])*(u[i][j][k]-u_old[i][j][k]));
	} else {
		value = 0.0;
	}
    value = warpReduceSum(value); 
    if (threadIdx.x % warpSize == 0){ // other idx?
		 atomicAdd(d, value);
	} 
}

__global__ 
void jacobi_reduction_presum(double ***u, double ***u_old, double ***f, int N, double delta, double *d) 
{ 
	double value = 0.0;
	idx_i = blockIdx.z * blockDim.z + threadIdx.z+1;
	idx_j = blockIdx.y * blockDim.y + threadIdx.y+1;
	idx_k = blockIdx.x * blockDim.x + threadIdx.x+1; 
    //double value = 0; 
    for (int i = idx_i; i < N+1; i += blockDim.z * gridDim.z){
		for (int j = idx_j; j < N+1; i += blockDim.y * gridDim.y){
			for (int k = idx_; k < N+1; i += blockDim.x * gridDim.x){
				tmpi = (u_old[i-1][j][k] + u_old[i+1][j][k]);
				tmpj = (u_old[i][j-1][k] + u_old[i][j+1][k]);
				tmpk = (u_old[i][j][k-1] + u_old[i][j][k+1]);
				u[i][j][k] = (tmpi + tmpj + tmpk + delta*f[i][j][k]) / 6.0;
				value += sqrt((u[i][j][k]-u_old[i][j][k])*(u[i][j][k]-u_old[i][j][k]));
			}
		}
	}  
    value = blockReduceSum(value); 
    if (threadIdx.x == 0){
		atomicAdd(d, value); 
	} 
}


__global__
void
jacobi_kernel(double ***u, double ***u_old, double ***f, int N, double delta) {
    int i, j, k;
	double tmpi, tmpj, tmpk;
	i = blockIdx.z * blockDim.z + threadIdx.z+1;
	j = blockIdx.y * blockDim.y + threadIdx.y+1;
	k = blockIdx.x * blockDim.x + threadIdx.x+1;
	
	if (i < N+1 && j < N+1 && k < N+1){
		tmpi = (u_old[i-1][j][k] + u_old[i+1][j][k]);
		tmpj = (u_old[i][j-1][k] + u_old[i][j+1][k]);
		tmpk = (u_old[i][j][k-1] + u_old[i][j][k+1]);
		u[i][j][k] = (tmpi + tmpj + tmpk + delta*f[i][j][k]) / 6.0;
	}
}

int
jacobi(double ***u_d, double ***u_old_d, double ***f_d, double ***u_h, double ***u_old_h, double ***f_h, int N, double delta, int iter_max, int NUM_BLOCKS, int THREADS_PER_BLOCK) {
	double*** temp;
	int k = 0;
    double d = 10000000.0;
	double tolerance = 1.0; 
	dim3 dimBlock(10,10,10); //Threads per block
    dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y,1); // Block in grid
	while(d>tolerance && k<iter_max)
    {
        // Execute kernel function
		jacobi_kernel<<<dimGrid,dimBlock>>>(u_d,u_old_d,f_d,N,delta);        
		checkCudaErrors(hipDeviceSynchronize());
		//  #Comment out when benchmarking!!#
        if ((k % 100) == 0)
		{   
			transfer_3d(u_h,u_d,N+2,N+2,N+2,hipMemcpyDeviceToHost);
			transfer_3d(u_old_h,u_old_d,N+2,N+2,N+2,hipMemcpyDeviceToHost);
            d = frobenius(u_h,u_old_h,N);
			printf("%i  %.5f\n", k, d);
        }
        //  #Comment out when benchmarking!!#
        temp = u_old_d;
        u_old_d = u_d;
        u_d  = temp;
        k+=1;
    }
	return k;
}
