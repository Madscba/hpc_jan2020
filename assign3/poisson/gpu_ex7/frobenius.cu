
#include <hip/hip_runtime.h>
#include <math.h>
double frobenius(double ***u, double ***u_old, int N){
    int i, j, k;
    double sum=0.0;
    double dist;
    for (i = 1; i < N+1; i++) 
	{
		for (j = 1; j < N+1; j++)   
		{
			for (k = 1; k < N+1; k++) 
			{   
                dist = u[i][j][k] - u_old[i][j][k];
                sum += dist*dist;
            }
        }
    }
    return sqrt(sum);

}
