#include "hip/hip_runtime.h"
/* main.c - Poisson problem in 3D
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include "print.h"
#include "alloc3d.h"
#include "alloc3d_gpu.h"
#include "transfer3d_gpu.h"
#include "matrix_init.h"
#include "matrix_overwrite.h"

#ifdef _JACOBI
#include <jacobi.h>
#endif

int
main(int argc, char *argv[]) {

    int N;                  // Dimension N x N x N.

    // Wake up GPU from power save state.
    //printf("Warming up device %i ... \n", device); fflush(stdout);
    hipSetDevice(0);           // Set the device to 0 or 1.
    hipDeviceEnablePeerAccess(1, 0); //Give read access to GPU 1
    double *dummy_d;
    hipMalloc((void**)&dummy_d, 0);

    double 	***u_h = NULL;
    double 	***u_old_h = NULL;
    double 	***f_h = NULL;
    double*** temp;
    int NUM_BLOCKS, THREADS_PER_BLOCK;


    int 	iter_max = 1000;
    double	start_T = 16.0;
    int		output_type = 1;
    char	*output_prefix = "poisson_res";
    char        *output_ext    = "";
    char	output_filename[FILENAME_MAX];
    int     lats;
    double  ts,te, mlups;


    /* get the paramters from the command line */
    N         = atoi(argv[1]);	// grid size
    iter_max  = atoi(argv[2]);  // max. no. of iterations
    NUM_BLOCKS  = atoi(argv[3]);  // no. of blocks
    THREADS_PER_BLOCK  = atoi(argv[4]);  // no. of threads per block
    if (argc == 6) {
    output_type = atoi(argv[5]);  // ouput type
    }

    const long nElms = (N+2) * (N+2) * (N+2); // Number of elements.

    // Allocate 3d array in host memory.
    if ( (u_h = d_malloc_3d(N+2, N+2, N+2)) == NULL ) {
        perror("array u: allocation failed");
        exit(-1);
    }
    if ( (u_old_h = d_malloc_3d(N+2, N+2, N+2)) == NULL ) {
        perror("array u: allocation failed");
        exit(-1);
    }
    if ( (f_h = d_malloc_3d(N+2, N+2, N+2)) == NULL ) {
        perror("array u: allocation failed");
        exit(-1);
    }
    
    double delta_sqr = (2/(N+2))*(2/(N+2));
    // Init u and f
    init_mat(N,start_T,f_h,u_old_h);
    init_bounds(N+2,20, 0, u_old_h);


    // Allocate 3d array on device 0 memory
    if ( (u_d0 = d_malloc_3d_gpu((N+2) / 2, N+2, N+2)) == NULL ) {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }
    if ( (u_old_d0 = d_malloc_3d_gpu((N+2) / 2, N+2, N+2)) == NULL ) {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }
    if ( (f_d0 = d_malloc_3d_gpu((N+2) / 2, N+2, N+2)) == NULL ) {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }

    // Transfer top part to device 0.
    transfer_3d_from_1d(u_d0, u_h[0][0], (N+2) / 2, N+2, N+2, hipMemcpyHostToDevice);
    transfer_3d_from_1d(u_old_d0, u_old_h[0][0], (N+2) / 2, N+2, N+2, hipMemcpyHostToDevice);
    transfer_3d_from_1d(f_d0, f_h[0][0], (N+2) / 2, N+2, N+2, hipMemcpyHostToDevice);


    double 	***u_d1 = NULL;
    double 	***u_old_d1 = NULL;
    double 	***f_d1 = NULL;
    hipSetDevice(1);
	hipDeviceEnablePeerAccess(0, 0); //Give read access to GPU 0
    // Allocate 3d array of half size in device 1 memory.
    if ( (u_d1 = d_malloc_3d_gpu((N+2) / 2, N+2, N+2)) == NULL ) {
        perror("array u_d1: allocation on gpu failed");
        exit(-1);
    }
    if ( (u_old_d1 = d_malloc_3d_gpu((N+2) / 2, N+2, N+2)) == NULL ) {
        perror("array u_d1: allocation on gpu failed");
        exit(-1);
    }
    if ( (f_d1 = d_malloc_3d_gpu((N+2) / 2, N+2, N+2)) == NULL ) {
        perror("array u_d1: allocation on gpu failed");
        exit(-1);
    }

    // Transfer bottom part to device 1.
    transfer_3d_from_1d(u_d1, u_h[0][0] + nElms / 2, (N+2) / 2, N+2, N+2, hipMemcpyHostToDevice);
    transfer_3d_from_1d(u_old_d1, u_old_h[0][0] + nElms / 2, (N+2) / 2, N+2, N+2, hipMemcpyHostToDevice);
    transfer_3d_from_1d(f_d1, f_h[0][0] + nElms / 2, (N+2) / 2, N+2, N+2, hipMemcpyHostToDevice);


    dim3 dimGrid(N/2,N/2,1); // 4096 blocks in total 
    dim3 dimBlock(N,N,1);// 256 threads per block

    int k = 0;
    // Loop until we meet stopping criteria
    ts = omp_get_wtime();
    while(k<iter_max)
    {
        #ifdef _JACOBI
        // Execute kernel function
        jacobi<<<dimGrid,dimBlock>>>(u_d0,u_old_d0,f_d0,N,delta_sqr);
        jacobi<<<dimGrid,dimBlock>>>(u_d1,u_old_d1,f_d1,N,delta_sqr);
        checkCudaErrors(hipDeviceSynchronize());
        #endif
        temp = u_old_d;
        u_old_d = u_d;
        u_d  = temp;
        k+=1;
    }
    te = omp_get_wtime();
    
    // Transfer back top part
    transfer_3d(u_h,u_d0,N+2,N+2,N+2,hipMemcpyDeviceToHost);
    transfer_3d(u_h,u_d0,N+2,N+2,N+2,hipMemcpyDeviceToHost);

    // dump  results if wanted 
    switch(output_type) {
    case 0:
        // no output at all
        break;
    case 1:
        lats = N*N*N;
        mlups = (double) lats*k/((te-ts)*1000*1000);
        printf("%d %.5f %.5f %d \n",N,mlups, te-ts, omp_get_max_threads());
        break;
        
    output_ext = ".vtk";
    sprintf(output_filename, "%s_%d%s", output_prefix, N, output_ext);
    fprintf(stderr, "Write VTK file to %s: ", output_filename);
    print_vtk(output_filename, N+2, u_h);
    // de-allocate memory
    free(u_h);
    free(u_old_h);
    free(f_h);
    free_gpu(u_d);
    free_gpu(u_old_d);
    free_gpu(f_d);
    return(0);
    }
}
