#include "hip/hip_runtime.h"
/* main.c - Poisson problem in 3D
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include "frobenius.h"
#include "alloc3d.h"
#include "alloc3d_gpu.h"
#include "transfer3d_gpu.h"
#include "matrix_init.h"
#include "matrix_overwrite.h"
#include "jacobi.h"

int
main(int argc, char *argv[]) {

    int N;                  // Dimension N x N x N.

    // Wake up GPU from power save state.
    //printf("Warming up device %i ... \n", device); fflush(stdout);
    hipSetDevice(0);           // Set the device to 0 or 1.
    hipDeviceEnablePeerAccess(1, 0); //Give read access to GPU 1
    double *dummy_d;
    hipMalloc((void**)&dummy_d, 0);

    double 	***u_h = NULL;
    double 	***u_old_h = NULL;
    double 	***f_h = NULL;
    double 	***u_d0 = NULL;
    double 	***u_old_d0 = NULL;
    double 	***f_d0 = NULL;
    double*** temp0;
    double*** temp1;
    int NUM_BLOCKS, THREADS_PER_BLOCK;


    int 	iter_max = 1000;
    double	start_T = 16.0;
    int		output_type = 1;
    char	*output_prefix = "poisson_res";
    char        *output_ext    = "";
    char	output_filename[FILENAME_MAX];
    int     lats;
    double  ts,te, mlups;


    /* get the paramters from the command line */
    N         = atoi(argv[1]);	// grid size
    iter_max  = atoi(argv[2]);  // max. no. of iterations
    NUM_BLOCKS  = atoi(argv[3]);  // no. of blocks
    THREADS_PER_BLOCK  = atoi(argv[4]);  // no. of threads per block
    if (argc == 6) {
    output_type = atoi(argv[5]);  // ouput type
    }

    const long nElms = (N+2) * (N+2) * (N+2); // Number of elements.

    // Allocate 3d array in host memory.
    if ( (u_h = d_malloc_3d(N+2, N+2, N+2)) == NULL ) {
        perror("array u: allocation failed");
        exit(-1);
    }
    if ( (u_old_h = d_malloc_3d(N+2, N+2, N+2)) == NULL ) {
        perror("array u: allocation failed");
        exit(-1);
    }
    if ( (f_h = d_malloc_3d(N+2, N+2, N+2)) == NULL ) {
        perror("array u: allocation failed");
        exit(-1);
    }
    
    double delta_sqr = (2/(N+2))*(2/(N+2));
    // Init u and f
    init_mat(N,start_T,f_h,u_h);
    init_mat(N,start_T,f_h,u_old_h);
    init_bounds(N+2,20, 0, u_old_h);


    // Allocate 3d array on device 0 memory
    if ( (u_d0 = d_malloc_3d_gpu((N+2) / 2, N+2, N+2)) == NULL ) {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }
    if ( (u_old_d0 = d_malloc_3d_gpu((N+2) / 2, N+2, N+2)) == NULL ) {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }
    if ( (f_d0 = d_malloc_3d_gpu((N+2) / 2, N+2, N+2)) == NULL ) {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }

    // Transfer top part to device 0.
    transfer_3d_from_1d(u_d0, u_h[0][0], (N+2) / 2, N+2, N+2, hipMemcpyHostToDevice);
    transfer_3d_from_1d(u_old_d0, u_old_h[0][0], (N+2) / 2, N+2, N+2, hipMemcpyHostToDevice);
    transfer_3d_from_1d(f_d0, f_h[0][0], (N+2) / 2, N+2, N+2, hipMemcpyHostToDevice);


    double 	***u_d1 = NULL;
    double 	***u_old_d1 = NULL;
    double 	***f_d1 = NULL;
    hipSetDevice(1);
	hipDeviceEnablePeerAccess(0, 0); //Give read access to GPU 0
    // Allocate 3d array of half size in device 1 memory.
    if ( (u_d1 = d_malloc_3d_gpu((N+2) / 2, N+2, N+2)) == NULL ) {
        perror("array u_d1: allocation on gpu failed");
        exit(-1);
    }
    if ( (u_old_d1 = d_malloc_3d_gpu((N+2) / 2, N+2, N+2)) == NULL ) {
        perror("array u_d1: allocation on gpu failed");
        exit(-1);
    }
    if ( (f_d1 = d_malloc_3d_gpu((N+2) / 2, N+2, N+2)) == NULL ) {
        perror("array u_d1: allocation on gpu failed");
        exit(-1);
    }

    // Transfer bottom part to device 1.
    transfer_3d_from_1d(u_d1, u_h[0][0] + nElms / 2, (N+2) / 2, N+2, N+2, hipMemcpyHostToDevice);
    transfer_3d_from_1d(u_old_d1, u_old_h[0][0] + nElms / 2, (N+2) / 2, N+2, N+2, hipMemcpyHostToDevice);
    transfer_3d_from_1d(f_d1, f_h[0][0] + nElms / 2, (N+2) / 2, N+2, N+2, hipMemcpyHostToDevice);


    dim3 dimGrid(N/2,N/2,1); // 4096 blocks in total 
    dim3 dimBlock(N,N,1);// 256 threads per block

    int k = 0;
    double d = 0.0;
    // Loop until we meet stopping criteria
    ts = omp_get_wtime();
    k = jacobi(u_d0, u_old_d0, f_d0, u_d1, u_old_d1, f_d1, u_h, u_old_h, f_h, N, delta_sqr, iter_max);
    te = omp_get_wtime();
    
    // Transfer back top part
    transfer_3d(u_h,u_d0,N+2,N+2,N+2,hipMemcpyDeviceToHost);
    transfer_3d(u_h,u_d1,N+2,N+2,N+2,hipMemcpyDeviceToHost);

    // dump  results if wanted 
    switch(output_type) {
    case 0:
        // no output at all
        break;
    case 1:
        lats = N*N*N;
        mlups = (double) lats*k/((te-ts)*1000*1000);
        printf("%d %.5f %.5f %d \n",N,mlups, te-ts, omp_get_max_threads());
        break;

    // de-allocate memory
    free(u_h);
    free(u_old_h);
    free(f_h);
    free_gpu(u_d0);
    free_gpu(u_old_d0);
    free_gpu(f_d0);
    free_gpu(u_d1);
    free_gpu(u_old_d1);
    free_gpu(f_d1);
    return(0);
    }
}
