#include "hip/hip_runtime.h"
/* jacobi.c - Poisson problem in 3d
 * 
 */
#include <math.h>
#include <float.h>
#include "frobenius.h"
#include <hip/hip_runtime_api.h>
#include "transfer3d_gpu.h"

__global__
void
jacobi_kernel(double ***u, double ***u_old, double ***f, int N, double delta) {
    int i, j, k;
	double tmpi, tmpj, tmpk;
	i = blockIdx.x * blockDim.x * gridDim.x + threadIdx.x+1;
	j = blockIdx.y * blockDim.y * gridDim.y  + threadIdx.y+1;
	k = blockIdx.z * blockDim.z * gridDim.z  + threadIdx.z+1;
	if (i < N+1 && j < N+1 && k < N+1){
		tmpi = (u_old[i-1][j][k] + u_old[i+1][j][k]);
		tmpj = (u_old[i][j-1][k] + u_old[i][j+1][k]);
		tmpk = (u_old[i][j][k-1] + u_old[i][j][k+1]);
		u[i][j][k] = (tmpi + tmpj + tmpk + delta*f[i][j][k]) / 6.0;
	}
}

int
jacobi(double ***u_d0, double ***u_old_d0, double ***f_d0,double ***u_d1, double ***u_old_d1, double ***f_d1, double ***u_h, double ***u_old_h, double ***f_h, int N, double delta, int iter_max) {
	double*** temp0;
	double*** temp1;
	int k = 0;
    double d = 0.0;
	dim3 dimBlock(10,10,10); //Threads per block
    dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y,1); // Block in grid
	while(k<iter_max)
    {
        #ifdef _JACOBI
        // Execute kernel function
        jacobi_kernel<<<dimGrid,dimBlock>>>(u_d0,u_old_d0,f_d0,N,delta_sqr);
        jacobi_kernel<<<dimGrid,dimBlock>>>(u_d1,u_old_d1,f_d1,N,delta_sqr);
        checkCudaErrors(hipDeviceSynchronize());
        #endif
        if ((k % 100) == 0)
		{   
    		transfer_3d(u_h,u_d0,N+2,N+2,N+2,hipMemcpyDeviceToHost);
    		transfer_3d(u_h,u_d1,N+2,N+2,N+2,hipMemcpyDeviceToHost);
    		transfer_3d(u_old_h,u_old_d0,N+2,N+2,N+2,hipMemcpyDeviceToHost);
    		transfer_3d(u_old_h,u_old_d1,N+2,N+2,N+2,hipMemcpyDeviceToHost);
            d = frobenius(u_h,u_old_h,N);
			printf("%i  %.5f\n", k, d);
        }
        temp0 = u_old_d0;
        u_old_d0 = u_d0;
        u_d0  = temp0;
        temp1 = u_old_d1;
        u_old_d1 = u_d1;
        u_d1  = temp1;
        k+=1;
    }
	return k;
}
