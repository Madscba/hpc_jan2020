#include "hip/hip_runtime.h"
/* jacobi.c - Poisson problem in 3d
 * 
 */
#include <math.h>
#include <float.h>
#include "frobenius.h"
#include <hip/hip_runtime_api.h>
#include "transfer3d_gpu.h"

__global__
void
jacobi_0(double ***u, double ***u_old_d0,double ***u_old_d1, double ***f, int N, double delta) {
    //Handle top part of matrix
    int i, j, k;
	double tmpi, tmpj, tmpk;
	i = blockIdx.z * blockDim.z + threadIdx.x+1; //We ignore the boundaries
	j = blockIdx.y * blockDim.y + threadIdx.y+1;
	k = blockIdx.x * blockDim.x + threadIdx.z+1;
	if (j < N+1 && k < N+1 && i == (N+2)/2-1 ) // i == (N+2)/2-1  if we are on bottom of the top half of the matrix
	{ 
		tmpi = (u_old_d0[i-1][j][k] + u_old_d1[0][j][k]); //We want to retrieve data from the 0'th index of the u_old_d1
		tmpj = (u_old_d0[i][j-1][k] + u_old_d0[i][j+1][k]);
		tmpk = (u_old_d0[i][j][k-1] + u_old_d0[i][j][k+1]);
		u[i][j][k] = (tmpi + tmpj + tmpk + delta*f[i][j][k]) / 6.0;
	} else if(i < (N+2)/2-2 && j < N+1 && k < N+1)
	{
		tmpi = (u_old_d0[i-1][j][k] + u_old_d0[i+1][j][k]);
		tmpj = (u_old_d0[i][j-1][k] + u_old_d0[i][j+1][k]);
		tmpk = (u_old_d0[i][j][k-1] + u_old_d0[i][j][k+1]);
		u[i][j][k] = (tmpi + tmpj + tmpk + delta*f[i][j][k]) / 6.0;
	}
}


__global__
void
jacobi_1(double ***u, double ***u_old_d0,double ***u_old_d1, double ***f, int N, double delta) {
	//Handle bottom part of matrix
    int i, j, k;
	double tmpi, tmpj, tmpk;
	i = blockIdx.z * blockDim.z + threadIdx.z; //we start in the middle of the hosts version of u.
	j = blockIdx.y * blockDim.y + threadIdx.y + 1; ////We ignore the boundaries
	k = blockIdx.x * blockDim.x + threadIdx.x + 1;
	if (j < N+1 && k < N+1 && i == 0) // i == 0 if we are on top of the bottom half of the matrix
	{ 
		tmpi = (u_old_d0[(N+2)/2-2][j][k] + u_old_d1[i+1][j][k]); //We want to retrieve data from the ((N+2)/2-2)'th index of the u_old_d0
		tmpj = (u_old_d1[i][j-1][k] + u_old_d1[i][j+1][k]);
		tmpk = (u_old_d1[i][j][k-1] + u_old_d1[i][j][k+1]);
		u[i][j][k] = (tmpi + tmpj + tmpk + delta*f[i][j][k]) / 6.0;
	} else if(i < (N+2)/2-1 && j < N+1 && k < N+1) // i < (N+2)/2-1 lets us ignore the bottom boundary of matrix
	{
		tmpi = (u_old_d1[i-1][j][k] + u_old_d1[i+1][j][k]);
		tmpj = (u_old_d1[i][j-1][k] + u_old_d1[i][j+1][k]);
		tmpk = (u_old_d1[i][j][k-1] + u_old_d1[i][j][k+1]);
		u[i][j][k] = (tmpi + tmpj + tmpk + delta*f[i][j][k]) / 6.0;
	}
}

int
jacobi(double ***u_d0, double ***u_old_d0, double ***f_d0,double ***u_d1, double ***u_old_d1, double ***f_d1, double ***u_h, double ***u_old_h, double ***f_h, int N, double delta, int iter_max) {
	double*** temp0;
	double*** temp1;
	int k = 0;
    double d = 0.0;
	//dim3 dimBlock(10,10,10); //Threads per block
    //dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y,1); // Block in grid
    
    const int BLOCK_SIZE = 10;
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid(ceil((double)(N+2)/BLOCK_SIZE/2),ceil((double)(N+2)/BLOCK_SIZE),ceil((double)(N+2)/BLOCK_SIZE));
	
	while(k<iter_max)
    {
        // Execute kernel function
		hipSetDevice(0);
		jacobi_0<<<dimGrid,dimBlock>>>(u_d0,u_old_d0,u_old_d1,f_d0,N,delta);
		hipSetDevice(1);
        jacobi_1<<<dimGrid,dimBlock>>>(u_d1,u_old_d0,u_old_d1,f_d1,N,delta);
        checkCudaErrors(hipDeviceSynchronize());
		hipSetDevice(0);
		checkCudaErrors(hipDeviceSynchronize());
        /*if ((k % 100) == 0)
		{   
			// Transfer back top part
			transfer_3d_to_1d(u_h[0][0],u_d0,(N+2)/2,(N+2),(N+2),hipMemcpyDeviceToHost);
			transfer_3d_to_1d(u_old_h[0][0],u_old_d0,(N+2)/2,(N+2),(N+2),hipMemcpyDeviceToHost);
            d = frobenius(u_h,u_old_h,N);
			printf("%i  %.5f\n", k, d);
        }*/
        temp0 = u_old_d0;
        u_old_d0 = u_d0;
        u_d0  = temp0;
        temp1 = u_old_d1;
        u_old_d1 = u_d1;
        u_d1  = temp1;
        k+=1;
    }
	return k;
}
