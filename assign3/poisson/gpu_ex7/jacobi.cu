/* jacobi.c - Poisson problem in 3d
 * 
 */

#include <hip/hip_runtime.h>
#include <math.h>
#include <float.h>

__global__
void
jacobi(double ***u, double ***u_old, double ***f, int N, double delta) {
    int i, j, k;
	double tmpi, tmpj, tmpk;
	i = blockIdx.x * blockDim.x * gridDim.x + threadIdx.x+1;
	j = blockIdx.y * blockDim.y * gridDim.y  + threadIdx.y+1;
	k = blockIdx.z * blockDim.z * gridDim.z  + threadIdx.z+1;
	if (i < N+1 && j < N+1 && k < N+1){
		tmpi = (u_old[i-1][j][k] + u_old[i+1][j][k]);
		tmpj = (u_old[i][j-1][k] + u_old[i][j+1][k]);
		tmpk = (u_old[i][j][k-1] + u_old[i][j][k+1]);
		u[i][j][k] = (tmpi + tmpj + tmpk + delta*f[i][j][k]) / 6.0;
	}
}
