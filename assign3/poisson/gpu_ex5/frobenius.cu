
#include <hip/hip_runtime.h>
#include <math.h>
__global__
void frobenius_kernel(double ***u, double ***u_old, int N,double sum){
    int i, j, k;
    double dist;
    for (i = 1; i < N+1; i++) 
	{
		for (j = 1; j < N+1; j++)   
		{
			for (k = 1; k < N+1; k++) 
			{   
                dist = u[i][j][k] - u_old[i][j][k];
                sum += dist*dist;
            }
        }
    }
}

double frobenius(double ***u, double ***u_old, int N){
    double sum=0.0;
	frobenius_kernel<<<1,1>>>(u,u_old,N,sum);
    return sqrt(sum);
}
