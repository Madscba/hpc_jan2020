#include "hip/hip_runtime.h"
#include <math.h>
__global__
void frobenius_kernel(double ***u, double ***u_old, int N,double sum){
    int i, j, k;
    double dist;
    for (i = 1; i < N+1; i++) 
	{
		for (j = 1; j < N+1; j++)   
		{
			for (k = 1; k < N+1; k++) 
			{   
                dist = u[i][j][k] - u_old[i][j][k];
                sum += dist*dist;
            }
        }
    }
}

double frobenius(double ***u, double ***u_old, int N){
    double sum_h;
    double sum_d;
    hipMalloc((void*)sum_d, sizeof(double));
	frobenius_kernel<<<1,1>>>(u,u_old,N,sum_d);
    hipMemcpy(sum_h, sum_d, sizeof(double), hipMemcpyDeviceToHost);
    return sqrt(sum_h);
}

