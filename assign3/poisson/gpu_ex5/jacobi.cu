#include "hip/hip_runtime.h"
/* jacobi.c - Poisson problem in 3d
 * 
 */
#include <math.h>
#include <float.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include "transfer3d_gpu.h"
#include "frobenius.h"
#include "alloc3d_gpu.h"


__global__
void
jacobi_kernel(double ***u, double ***u_old, double ***f, int N, double delta) {
    int i, j, k;
	double tmpi, tmpj, tmpk;
	printf("OLD0: %f NEW: %f \n",u_old[0][0][0],u[0][0][0]);
    for (i = 1; i < N+1; i++) 
	{
		for (j = 1; j < N+1; j++)   
		{
			for (k = 1; k < N+1; k++) 
			{	
				tmpi = (u_old[i-1][j][k] + u_old[i+1][j][k]);
				printf("i-1 %f i+1 %f tmpi %f sum %f",u_old[i-1][j][k],u_old[i+1][j][k],tmpi,u_old[i-1][j][k]+u_old[i+1][j][k]);
				tmpj = (u_old[i][j-1][k] + u_old[i][j+1][k]);
				printf(" j-1: %f j+1: %f ",u_old[i][j-1][k],u_old[i][j+1][k]);
				tmpk = (u_old[i][j][k-1] + u_old[i][j][k+1]);
				printf(" k-1: %f k+1: %f \n",u_old[i][j][k-1],u_old[i][j][k+1]);
				u[i][j][k] = (tmpi + tmpj + tmpk + delta*f[i][j][k]) / 6.0;
				printf("(%i,%i,%i) f: %f i %f j %f k %f u %f old %f \n",i,j,k,f[i][j][k],tmpi,tmpj,tmpk,u[i][j][k],u_old[i][j][k]);
			}
		}
	}
}

int
jacobi(double ***u_d, double ***u_old_d, double ***f_d, double ***u_h, double ***u_old_h, double ***f_h, int N, double delta, int iter_max) {
	double*** temp;
	int k = 0;
    double d = 0.0;

	while(k<iter_max)
    {
        // Execute kernel function
        jacobi_kernel<<<1,1>>>(u_d,u_old_d,f_d,N,delta);
        checkCudaErrors(hipDeviceSynchronize());
		//  #Comment out when benchmarking!!#
        if ((k % 1) == 0)
		{   
			transfer_3d(u_h,u_d,N+2,N+2,N+2,hipMemcpyDeviceToHost);
			transfer_3d(u_old_h,u_old_d,N+2,N+2,N+2,hipMemcpyDeviceToHost);
			printf("%f \n",u_h[1][2][2]); 
			printf("%f \n",u_old_h[1][2][2]);
            d = frobenius(u_h,u_old_h,N);
			printf("%i  %.5f\n", k, d);
        }
        //  #Comment out when benchmarking!!#
        temp = u_old_d;
        u_old_d = u_d;
        u_d  = temp;
        k+=1;
    }
	return k; 
}
