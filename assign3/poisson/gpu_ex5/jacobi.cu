#include "hip/hip_runtime.h"
/* jacobi.c - Poisson problem in 3d
 * 
 */
#include <math.h>
#include <float.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include "transfer3d_gpu.h"
#include "frobenius.h"
#include "alloc3d_gpu.h"


__global__
void
jacobi_kernel(double ***u, double ***u_old, double ***f, int N, double delta) {
    int i, j, k;
	double tmpi, tmpj, tmpk;
	printf("kernel %f \n",u[2][2][2]); 
	printf("kernel %f \n",u_old[2][2][2]);
    for (i = 1; i < N+1; i++) 
	{
		for (j = 1; j < N+1; j++)   
		{
			for (k = 1; k < N+1; k++) 
			{	
				tmpi = (u_old[i-1][j][k] + u_old[i+1][j][k]);
				tmpj = (u_old[i][j-1][k] + u_old[i][j+1][k]);
				tmpk = (u_old[i][j][k-1] + u_old[i][j][k+1]);
				u[i][j][k] = (tmpi + tmpj + tmpk + delta*f[i][j][k]) / 6.0;
			}
		}
	}
}

int
jacobi(double ***u_d, double ***u_old_d, double ***f_d, double ***u_h, double ***u_old_h, double ***f_h, int N, double delta, int iter_max) {
	double*** temp;
	int k = 0;
    double d = 0.0;

	while(k<iter_max)
    {
        // Execute kernel function
        jacobi_kernel<<<1,1>>>(u_d,u_old_d,f_d,N,delta);
        checkCudaErrors(hipDeviceSynchronize());
		//  #Comment out when benchmarking!!#
        if ((k % 100) == 0)
		{   
			transfer_3d(u_h,u_d,N+2,N+2,N+2,hipMemcpyDeviceToHost);
			transfer_3d(u_old_h,u_old_d,N+2,N+2,N+2,hipMemcpyDeviceToHost);
			printf("%f \n",u_h[2][2][2]); 
			printf("%f \n",u_old_h[2][2][2]);
            d = frobenius(u_h,u_old_h,N);
			printf("%i  %.5f\n", k, d);
        }
        //  #Comment out when benchmarking!!#
        temp = u_old_d;
        u_old_d = u_d;
        u_d  = temp;
        k+=1;
    }
	return k; 
}
