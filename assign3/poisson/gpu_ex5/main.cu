#include "hip/hip_runtime.h"
/* main.c - Poisson problem in 3D
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include "alloc3d.h"
#include "alloc3d_gpu.h"
#include "transfer3d_gpu.h"
#include "matrix_init.h"
#include "matrix_overwrite.h"

#ifdef _JACOBI
#include <jacobi.h>
#endif

int
main(int argc, char *argv[]) {

    int N;                  // Dimension N x N x N.
    const int device = 0;   // Set the device to 0 or 1.

    // Wake up GPU from power save state.
    //printf("Warming up device %i ... \n", device); fflush(stdout);
    hipSetDevice(device);           // Set the device to 0 or 1.
    double *dummy_d;
    hipMalloc((void**)&dummy_d, 0);

    double 	***u_h = NULL;
    double 	***u_old_h = NULL;
    double 	***f_h = NULL;
    double 	***u_d = NULL;
    double 	***u_old_d = NULL;
    double 	***f_d = NULL;
    double*** temp;
    int NUM_BLOCKS, THREADS_PER_BLOCK;


    int 	iter_max = 1000;
    double	start_T = 16.0;
    int		output_type = 1;
    char	*output_prefix = "poisson_res";
    char        *output_ext    = "";
    char	output_filename[FILENAME_MAX];
    int     lats;
    double  ts,te, mlups;


    /* get the paramters from the command line */
    N         = atoi(argv[1]);	// grid size
    iter_max  = atoi(argv[2]);  // max. no. of iterations
    NUM_BLOCKS  = atoi(argv[3]);  // no. of blocks
    THREADS_PER_BLOCK  = atoi(argv[4]);  // no. of threads per block
    if (argc == 6) {
    output_type = atoi(argv[5]);  // ouput type
    }

    const long nElms = N * N * N; // Number of elements.

    // Allocate 3d array in host memory.
    if ( (u_h = d_malloc_3d(N+2, N+2, N+2)) == NULL ) {
        perror("array u: allocation failed");
        exit(-1);
    }
    if ( (u_old_h = d_malloc_3d(N+2, N+2, N+2)) == NULL ) {
        perror("array u: allocation failed");
        exit(-1);
    }
    if ( (f_h = d_malloc_3d(N+2, N+2, N+2)) == NULL ) {
        perror("array u: allocation failed");
        exit(-1);
    }
    
    double delta_sqr = (2/(N+2))*(2/(N+2));
    // Init u and f
    init_mat(N,start_T,f_h,u_old_h);
    init_bounds(N+2,20, 0, u_old_h);


    // Allocate 3d array on device 0 memory
    if ( (u_d = d_malloc_3d_gpu(N+2, N+2, N+2)) == NULL ) {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }
    if ( (u_old_d = d_malloc_3d_gpu(N+2, N+2, N+2)) == NULL ) {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }
    if ( (f_d = d_malloc_3d_gpu(N+2, N+2, N+2)) == NULL ) {
        perror("array u_d0: allocation on gpu failed");
        exit(-1);
    }



    // Transfer to device 0.
    transfer_3d_from_1d(u_d, u_h[0][0], N+2, N+2, N+2, hipMemcpyHostToDevice);
    transfer_3d_from_1d(u_old_d, u_old_h[0][0], N+2, N+2, N+2, hipMemcpyHostToDevice);
    transfer_3d_from_1d(f_d, f_h[0][0], N+2, N+2, N+2, hipMemcpyHostToDevice);


    int k = 0;
    // Loop until we meet stopping criteria
    ts = omp_get_wtime();
    while(k<iter_max)
    {
        #ifdef _JACOBI
        // Execute kernel function
        jacobi<<<1,1>>>(u_d,u_old_d,f_d,N,delta_sqr);
        checkCudaErrors(hipDeviceSynchronize());
        #endif
        temp = u_old_d;
        u_old_d = u_d;
        u_d  = temp;
        k+=1;
    }
    te = omp_get_wtime();
    
    // Transfer back
    transfer_3d(u_h,u_d,N+2,N+2,N+2,hipMemcpyDeviceToHost);
   

    // dump  results if wanted 
    switch(output_type) {
    case 0:
        // no output at all
        break;
    case 1:
        lats = N*N*N;
        mlups = (double) lats*k/((te-ts)*1000*1000);
        printf("%d %.5f %.5f %d \n",N,mlups, te-ts, omp_get_max_threads());
        break;

    // de-allocate memory
    free(u_h);
    free(u_old_h);
    free(f_h);
    free_gpu(u_d);
    free_gpu(u_old_d);
    free_gpu(f_d);
    return(0);
    }
}
